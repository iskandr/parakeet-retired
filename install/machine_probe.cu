/*
 * Parakeet
 *
 * (c) 2009-2011 Eric Hielscher, Alex Rubinsteyn
 *
 * Machine Probe
 *
 * Utility for detecting main architectural characteristics of the given
 * computer for use in Parakeet's code optimization.
 * 
 * Outputs an XML file with the gathered information for use by the Parakeet
 * runtime.
 */

#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

typedef struct {
  int id;
  hipDeviceProp_t deviceProp;
  int *accessiblePeers;
  int numAccessiblePeers;
  int globalMemspace;
  float globalPeakBw;
} gpu_t;

typedef struct {
  int id;
  uint64_t bytes;
} memspace_t;

typedef struct {
  int srcId;
  int *dstIds;
  int numDsts;
  float bw;
} mem_xfer_bw_t;

void add_xfer_bw(mem_xfer_bw_t **bws, int numBws, int numDsts) {
  mem_xfer_bw_t *tmp =
      (mem_xfer_bw_t*)malloc((numBws + 1) * sizeof(mem_xfer_bw_t));

  if (*bws) {
    memcpy(tmp, *bws, numBws * sizeof(mem_xfer_bw_t));
    free(*bws);
  }

  *bws = tmp;
  (*bws)[numBws].dstIds = (int*)malloc(numDsts * sizeof(int));
  (*bws)[numBws].numDsts = numDsts;
}

void free_xfer_bws(mem_xfer_bw_t *bws, int numBws) {
  if (!bws) return;
  
  int i;
  for (i = 0; i < numBws; ++i) {
    free(bws[i].dstIds);
  }
  free(bws);
}

void chkError(int rslt, char *msg) {
  if (rslt != 0) {
    printf("%s: %d\n", msg, rslt);
    exit(1);
  }
}

double diff_timers(struct timeval *start, struct timeval *end) {
  double ret;

  if (end->tv_usec < start->tv_usec) {
    int nsec = (start->tv_usec - end->tv_usec) / 1000000 + 1;
    start->tv_usec -= 1000000 * nsec;
    start->tv_sec += nsec;
  }
  if (end->tv_usec - start->tv_usec > 1000000) {
    int nsec = (end->tv_usec - start->tv_usec) / 1000000;
    start->tv_usec += 1000000 * nsec;
    start->tv_sec -= nsec;
  }

  ret = (end->tv_sec - start->tv_sec) +
        (end->tv_usec - start->tv_usec) / 1000000.0;

  free(start);
  free(end);

  return ret;
}

struct timeval *gettime(void) {
  struct timeval *ret = (struct timeval*)(malloc(sizeof(struct timeval)));
  gettimeofday(ret, NULL);
  return ret;
}

// Assumes that the current GPU device is set
float time_ram_to_gpu_xfer(void *dev_data, void *ram_data, int data_size) {
  struct timeval *start, *end;
  float ram_gpu_bw = 0.0f;
  hipStreamSynchronize(0);
  start = gettime();
  chkError(hipMemcpy(dev_data, ram_data, data_size, hipMemcpyHostToDevice),
           "Couldn't copy data from RAM to GPU");
  hipStreamSynchronize(0);
  end = gettime();
  ram_gpu_bw += diff_timers(start, end);  
  return data_size / ram_gpu_bw / (1 << 30);
}  

int main(int argc, char **argv) {
  const int RAMID = 0;
  const int PINNEDID = 1;
  const int GPUOFFSET = 2;

  struct timeval *start, *end;
  int i, j;

  // Set up program parameters
  // TODO: We assume here that any GPU we're going to use has at least 128MB of
  //       global memory.  This may not actually be the case.  We probably want
  //       to parameterize this so as to scale to any memory size.
  int data_size = (16 << 20) * sizeof(int);
  char *outFilename = "parakeetconf.xml";
  int debug = 1;

  // Process command line args
  
  // Open output file
  FILE *outfile = fopen(outFilename, "w");
  if (!outfile) {
    printf("Couldn't open output file.\n");
    exit(1);
  }

  // Get number of GPU devices
  int numDevices;
  chkError(hipGetDeviceCount(&numDevices), "Couldn't get number of devices");
  if (numDevices > sizeof(int) * 8 - 1) {
    printf("Can't support more than %d devices\n", sizeof(int) * 8 - 1);
    exit(1);
  }
  
  // Create a gpu_t struct for each device
  gpu_t *gpus = (gpu_t*)malloc(numDevices * sizeof(gpu_t));
  
  // Create memspace structs for RAM and for each device
  memspace_t *memspaces =
      (memspace_t*)malloc((numDevices + 1) * sizeof(memspace_t));
  for (i = 0; i < numDevices + 1; ++i) {
    memspaces[i].id = i;
  }
  
  // Memory transfer structs for each valid memspace group
  int numBws = 0;
  mem_xfer_bw_t *bws = NULL;
  
  // Set up special RAM memspace
  // TODO: This probably is Ubuntu-specific; need to make it general.
  char *cmd = "awk '{if(NR==1){print $2}}' /proc/meminfo";
  FILE *cmdfile = popen(cmd, "r");
  if (!cmdfile) {
    printf("Unable to get RAM info.\n");
    exit(1);
  }
  char buffer[128];
  memset(buffer, 0, 128);
  if (!fgets(buffer, 128, cmdfile)) {
    printf("Unable to read RAM info from /proc/meminfo.\n");
    exit(1);
  }
  memspaces[RAMID].bytes = (uint64_t)atol(buffer);
  if (!memspaces[RAMID].bytes) {
    printf("Unable to convert RAM info to uint64_t.\n");
    exit(1);
  }
  pclose(cmdfile);
  
  // Allocate some memory for doing RAM <-> GPU transfers.
  int *ram_data = (int*)malloc(data_size);
  int *pinned_data;
  chkError(hipHostMalloc(&pinned_data, data_size),
           "Couldn't malloc pinned host mem");
  int **dev_datas = (int**)malloc(numDevices * sizeof(int*));
  
  // For each device, get the properties we're interested in
  for (i = 0; i < numDevices; ++i) {
    // Current memspace ID is i + 1, since RAM is 0
    int curId = i + GPUOFFSET;
    
    // Get device properties
    // TODO: Do we need to store this? Could just re-query every time.
    chkError(hipGetDeviceProperties(&gpus[i].deviceProp, i),
             "Couldn't get properties for device");
    
    // Store the calculated peak global memory b/w
    // TODO: Assumes that all GPUs use DDR, and so uses a x2 multiplier.
    //       If this ever changes, this won't be accurate.
    gpus[i].globalPeakBw =
        gpus[i].deviceProp.memoryClockRate * 2.0f / 1000000.0f *
        gpus[i].deviceProp.memoryBusWidth / 8.0f;

    if (debug) printf("GPU %d Theoretical Peak Global B/W: %f\n",
                      i, gpus[i].globalPeakBw);
    
    // Allocate some device memory space
    chkError(hipSetDevice(i), "Couldn't switch GPU devices");
    chkError(hipMalloc(&dev_datas[i], data_size),
             "Couldn't allocate GPU data");

    // Get peer access info
    gpus[i].numAccessiblePeers = 0;
    int canAccessPeer;
    for (j = 0; j < numDevices; ++j) {
      if (i != j) {
        chkError(hipDeviceCanAccessPeer(&canAccessPeer, i, j),
                 "Couldn't get peer access info");
        if (canAccessPeer) {
          gpus[i].numAccessiblePeers++;
          chkError(hipDeviceEnablePeerAccess(j, 0),
                   "Couldn't enable peer access");
        }
      }
    }
    gpus[i].accessiblePeers =
        (int*)malloc(gpus[i].numAccessiblePeers * sizeof(int));
    int cur = 0;
    for (j = 0; j < numDevices; ++j) {
      if (i != j) {
        chkError(hipDeviceCanAccessPeer(&canAccessPeer, i, j),
                 "Couldn't get peer access info");
        if (canAccessPeer) {
          gpus[i].accessiblePeers[cur++] = j;
          
          // Test P2P memory bandwidth and record 
          chkError(hipSetDevice(j), "Coudln't switch GPU devices");
          int *src_data;
          chkError(hipMalloc(&src_data, data_size),
                   "Couldn't allocate peer GPU data");
          chkError(hipSetDevice(i), "Couldn't switch GPU devices");
          
          float peer_bw = 0.0f;
          hipStreamSynchronize(0);
          start = gettime();
          chkError(hipMemcpyPeer(dev_datas[i], i, src_data, j, data_size),
                    "Couldn't copy data between peer devices");
          hipStreamSynchronize(0);
          end = gettime();
          peer_bw = data_size / diff_timers(start, end) / (1 << 30);
          add_xfer_bw(&bws, numBws, 1);
          bws[numBws].srcId = curId;
          bws[numBws].dstIds[0] = j + GPUOFFSET;
          bws[numBws].bw = peer_bw;
          numBws++;
          
          if (debug) printf("P2P transfer from %d to %d: %f\n", j, i, peer_bw);
          
          chkError(hipSetDevice(j), "Couldn't switch GPU devices");
          chkError(hipFree(src_data), "Couldn't free peer GPU data");
          chkError(hipSetDevice(i), "Couldn't switch GPU devices");
        }
      }
    }
  }
  
  // Test RAM <-> devices B/W for every combination of devices
  int numSets = 1 << numDevices;
  int devs;
  for (devs = 1; devs < numSets; ++devs) {
    // Test RAM <-> GPUs bw
    int curNumDevs = __builtin_popcount(devs);
    add_xfer_bw(&bws, numBws, curNumDevs);
    bws[numBws].srcId = RAMID;

    int curDev = 0;
    for (i = 0; i < numDevices; ++i) {
      chkError(hipSetDevice(i), "Couldn't set device");
      hipStreamSynchronize(0);
    }
    start = gettime();
    for (i = 0; i < numDevices; ++i) {
      if ((1 << i) & devs) {
        chkError(hipSetDevice(i), "Couldn't set device");
        chkError(hipMemcpy(dev_datas[i], ram_data, data_size,
                            hipMemcpyHostToDevice),
                 "Couldn't copy data from RAM to GPU");
        bws[numBws].dstIds[curDev++] = i;
      }
    }
    for (i = 0; i < numDevices; ++i) {
      chkError(hipSetDevice(i), "Couldn't set device");
      hipStreamSynchronize(0);
    }
    end = gettime();
    bws[numBws].bw =
        data_size * curNumDevs / diff_timers(start, end) / (1 << 30);
    numBws++;
    
    if (debug) {
      if (curNumDevs == 1) {
        printf("RAM to GPU %d B/W: %f\n",
               bws[numBws - 1].dstIds[0],
               bws[numBws - 1].bw);
      } else {
        printf("RAM to %d GPUs B/W: %f\n", curNumDevs, bws[numBws - 1].bw);
      }
    }
    
    // Test Pinned RAM <-> GPU bw
    /*
    add_xfer_bw(&bws, numBws, 1);
    bws[numBws].srcId = PINNEDID;
    bws[numBws].dstIds[0] = curId;
    bws[numBws].bw = time_ram_to_gpu_xfer(dev_data, pinned_data, data_size);
    numBws++;
    
    if (debug) printf("Pinned RAM to GPU %d B/W: %f\n", i, bws[numBws - 1].bw);
    */
  }    

  free(ram_data);
  hipFree(pinned_data);
  for (i = 0; i < numDevices; ++i) {
    free(gpus[i].accessiblePeers);
    hipFree(dev_datas[i]);
  }
  free(gpus);
  free_xfer_bws(bws, numBws);

  return 0;
}
