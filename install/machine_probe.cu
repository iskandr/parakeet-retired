/*
 * Parakeet
 *
 * (c) 2009-2011 Eric Hielscher, Alex Rubinsteyn
 *
 * Machine Probe
 *
 * Utility for detecting main architectural characteristics of the given
 * computer for use in Parakeet's code optimization.
 * 
 * Outputs an XML file with the gathered information for use by the Parakeet
 * runtime.
 */

#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

// Data structures to hold the machine characteristics
typedef struct {
  int id;
  hipDeviceProp_t deviceProp;
  int *accessiblePeers;
  int numAccessiblePeers;
  int globalMemspace;
  float globalPeakBw;
} gpu_t;

typedef struct {
  int id;
  uint64_t bytes;
} memspace_t;

typedef struct {
  int srcId;
  int *dstIds;
  int numDsts;
  float bw;
} mem_xfer_bw_t;

// Helper functions to add and free the memory transfer b/w structs
void add_xfer_bw(mem_xfer_bw_t **bws, int numBws, int numDsts) {
  mem_xfer_bw_t *tmp =
      (mem_xfer_bw_t*)malloc((numBws + 1) * sizeof(mem_xfer_bw_t));

  if (*bws) {
    memcpy(tmp, *bws, numBws * sizeof(mem_xfer_bw_t));
    free(*bws);
  }

  *bws = tmp;
  (*bws)[numBws].dstIds = (int*)malloc(numDsts * sizeof(int));
  (*bws)[numBws].numDsts = numDsts;
}

void free_xfer_bws(mem_xfer_bw_t *bws, int numBws) {
  if (!bws) return;
  
  int i;
  for (i = 0; i < numBws; ++i) {
    free(bws[i].dstIds);
  }
  free(bws);
}

void chkError(int rslt, char *msg) {
  if (rslt != 0) {
    printf("%s: %d\n", msg, rslt);
    exit(1);
  }
}

// Timer helper functions
double diff_timers(struct timeval *start, struct timeval *end) {
  double ret;

  if (end->tv_usec < start->tv_usec) {
    int nsec = (start->tv_usec - end->tv_usec) / 1000000 + 1;
    start->tv_usec -= 1000000 * nsec;
    start->tv_sec += nsec;
  }
  if (end->tv_usec - start->tv_usec > 1000000) {
    int nsec = (end->tv_usec - start->tv_usec) / 1000000;
    start->tv_usec += 1000000 * nsec;
    start->tv_sec -= nsec;
  }

  ret = (end->tv_sec - start->tv_sec) +
        (end->tv_usec - start->tv_usec) / 1000000.0;

  free(start);
  free(end);

  return ret;
}

struct timeval *gettime(void) {
  struct timeval *ret = (struct timeval*)(malloc(sizeof(struct timeval)));
  gettimeofday(ret, NULL);
  return ret;
}

// Global state
const int RAMID = 0;
const int PINNEDID = 1;
const int GPUOFFSET = 2;
mem_xfer_bw_t *bws = NULL;
int numBws = 0;
gpu_t *gpus = NULL;
int numDevices = 0;
int *ram_data;
int *pinned_data;
int **dev_datas;
FILE *outfile;
struct timeval *start, *end;
int debug;

// Test the memory transfer bandwidth between the src and all the destination
// devices in the bitmask devs
void test_mem_xfer_bw(int *src_data, int data_size, int srcId, int devs,
                      char *src_name) {
  int curNumDevs = __builtin_popcount(devs);
  add_xfer_bw(&bws, numBws, curNumDevs);
  bws[numBws].srcId = srcId;

  int curDev = 0;
  int i;
  for (i = 0; i < numDevices; ++i) {
    chkError(hipSetDevice(i), "Couldn't set device");
    hipStreamSynchronize(0);
  }
  start = gettime();
  for (i = 0; i < numDevices; ++i) {
    if ((1 << i) & devs) {
      chkError(hipSetDevice(i), "Couldn't set device");
      chkError(hipMemcpy(dev_datas[i], src_data, data_size,
                          hipMemcpyHostToDevice),
                "Couldn't copy data from RAM to GPU");
      bws[numBws].dstIds[curDev++] = i + GPUOFFSET;
    }
  }
  for (i = 0; i < numDevices; ++i) {
    chkError(hipSetDevice(i), "Couldn't set device");
    hipStreamSynchronize(0);
  }
  end = gettime();
  bws[numBws].bw =
      data_size * curNumDevs / diff_timers(start, end) / (1 << 30);
  numBws++;
  
  if (debug) {
    if (curNumDevs == 1) {
      printf("%s to GPU %d B/W: %f\n",
             src_name, bws[numBws - 1].dstIds[0], bws[numBws - 1].bw);
    } else {
      printf("%s to %d GPUs B/W: %f\n",
             src_name, curNumDevs, bws[numBws - 1].bw);
    }
  }
}

int main(int argc, char **argv) {
  int i, j;

  // Set up program parameters
  // TODO: We assume here that any GPU we're going to use has at least 128MB of
  //       global memory.  This may not actually be the case.  We probably want
  //       to parameterize this so as to scale to any memory size.
  int data_size = (16 << 20) * sizeof(int);
  char *outFilename = "parakeetconf.xml";
  debug = 1;

  // Process command line args
  
  // Open output file
  outfile = fopen(outFilename, "w");
  if (!outfile) {
    printf("Couldn't open output file.\n");
    exit(1);
  }

  // Get number of GPU devices
  chkError(hipGetDeviceCount(&numDevices), "Couldn't get number of devices");
  if (numDevices > sizeof(int) * 8 - 1) {
    printf("Can't support more than %d devices\n", sizeof(int) * 8 - 1);
    exit(1);
  }
  
  // Create a gpu_t struct for each device
  gpus = (gpu_t*)malloc(numDevices * sizeof(gpu_t));
  
  // Create memspace structs for RAM, pinned RAM and for each device
  memspace_t *memspaces =
      (memspace_t*)malloc((numDevices + GPUOFFSET) * sizeof(memspace_t));
  for (i = 0; i < numDevices + GPUOFFSET; ++i) {
    memspaces[i].id = i;
  }
  
  // Set up special RAM memspace
  // TODO: This probably is Ubuntu-specific; need to make it general.
  char *cmd = "awk '{if(NR==1){print $2}}' /proc/meminfo";
  FILE *cmdfile = popen(cmd, "r");
  if (!cmdfile) {
    printf("Unable to get RAM info.\n");
    exit(1);
  }
  char buffer[128];
  memset(buffer, 0, 128);
  if (!fgets(buffer, 128, cmdfile)) {
    printf("Unable to read RAM info from /proc/meminfo.\n");
    exit(1);
  }
  memspaces[RAMID].bytes = (uint64_t)atol(buffer) * 1024;
  if (!memspaces[RAMID].bytes) {
    printf("Unable to convert RAM info to uint64_t.\n");
    exit(1);
  }
  memspaces[PINNEDID].bytes = memspaces[RAMID].bytes;
  pclose(cmdfile);
  
  // Allocate some memory for doing RAM <-> GPU transfers.
  ram_data = (int*)malloc(data_size);
  chkError(hipHostMalloc(&pinned_data, data_size),
           "Couldn't malloc pinned host mem");
  dev_datas = (int**)malloc(numDevices * sizeof(int*));
  
  // For each device, get the properties we're interested in
  for (i = 0; i < numDevices; ++i) {
    // Get device properties
    // TODO: Do we need to store this? Could just re-query every time.
    chkError(hipGetDeviceProperties(&gpus[i].deviceProp, i),
             "Couldn't get properties for device");
    
    // Take into account that RAM = 0 and PinnedRam = 1
    gpus[i].globalMemspace = i + GPUOFFSET;
    memspaces[i+GPUOFFSET].bytes = gpus[i].deviceProp.totalGlobalMem;
    
    // Store the calculated peak global memory b/w
    // TODO: Assumes that all GPUs use DDR, and so uses a x2 multiplier.
    //       If this ever changes, this won't be accurate.
    gpus[i].globalPeakBw =
        gpus[i].deviceProp.memoryClockRate * 2.0f / 1000000.0f *
        gpus[i].deviceProp.memoryBusWidth / 8.0f;

    if (debug) printf("GPU %d Theoretical Peak Global B/W: %f\n",
                      i, gpus[i].globalPeakBw);
    
    // Allocate some device memory space
    chkError(hipSetDevice(i), "Couldn't switch GPU devices");
    chkError(hipMalloc(&dev_datas[i], data_size),
             "Couldn't allocate GPU data");

    // Get peer access info
    gpus[i].numAccessiblePeers = 0;
    int canAccessPeer;
    for (j = 0; j < numDevices; ++j) {
      if (i != j) {
        chkError(hipDeviceCanAccessPeer(&canAccessPeer, i, j),
                 "Couldn't get peer access info");
        if (canAccessPeer) {
          gpus[i].numAccessiblePeers++;
          chkError(hipDeviceEnablePeerAccess(j, 0),
                   "Couldn't enable peer access");
        }
      }
    }
    gpus[i].accessiblePeers =
        (int*)malloc(gpus[i].numAccessiblePeers * sizeof(int));
    int cur = 0;
    for (j = 0; j < numDevices; ++j) {
      if (i != j) {
        chkError(hipDeviceCanAccessPeer(&canAccessPeer, i, j),
                 "Couldn't get peer access info");
        if (canAccessPeer) {
          gpus[i].accessiblePeers[cur++] = j;
          
          // Test P2P memory bandwidth and record 
          chkError(hipSetDevice(j), "Coudln't switch GPU devices");
          int *src_data;
          chkError(hipMalloc(&src_data, data_size),
                   "Couldn't allocate peer GPU data");
          chkError(hipSetDevice(i), "Couldn't switch GPU devices");
          
          float peer_bw = 0.0f;
          hipStreamSynchronize(0);
          start = gettime();
          chkError(hipMemcpyPeer(dev_datas[i], i, src_data, j, data_size),
                    "Couldn't copy data between peer devices");
          hipStreamSynchronize(0);
          end = gettime();
          peer_bw = data_size / diff_timers(start, end) / (1 << 30);
          add_xfer_bw(&bws, numBws, 1);
          bws[numBws].srcId = gpus[i].globalMemspace;
          bws[numBws].dstIds[0] = j + GPUOFFSET;
          bws[numBws].bw = peer_bw;
          numBws++;
          
          if (debug) printf("P2P transfer from %d to %d: %f\n", j, i, peer_bw);
          
          chkError(hipSetDevice(j), "Couldn't switch GPU devices");
          chkError(hipFree(src_data), "Couldn't free peer GPU data");
          chkError(hipSetDevice(i), "Couldn't switch GPU devices");
        }
      }
    }
  }
  
  // Test RAM <-> devices B/W for every combination of devices
  int numSets = 1 << numDevices;
  int devs;
  for (devs = 1; devs < numSets; ++devs) {
    // Test RAM <-> GPUs bw
    test_mem_xfer_bw(ram_data, data_size, RAMID, devs, "RAM");
    
    // Test Pinned RAM <-> GPU bw
    test_mem_xfer_bw(pinned_data, data_size, PINNEDID, devs, "Pinned RAM");
  }
  
  // Output XML file with the collected data
  int outLevel = 0;
  fprintf(outfile, "<Machine>\n");
  outLevel++;
  for (i = 0; i < numDevices; ++i) {
    fprintf(outfile, "%*s<GPU>\n", outLevel++, "");
    
    // Print out the contents of the CUDA device properties struct
    hipDeviceProp_t curProp = gpus[i].deviceProp;
    fprintf(outfile, "%*s<Id>%d</Id>\n", outLevel, "", i);
    fprintf(outfile, "%*s<DeviceName>%s</DeviceName>\n", outLevel, "",
            curProp.name);
    fprintf(outfile, "%*s<TotalGlobalMemory>%ld</TotalGlobalMemory>\n",
            outLevel, "", curProp.totalGlobalMem);
    fprintf(outfile, "%*s<SharedMemPerSM>%ld</SharedMemPerSM>\n",
            outLevel, "", curProp.sharedMemPerBlock);
    fprintf(outfile, "%*s<RegsPerBlock>%d</RegsPerBlock>\n",
            outLevel, "", curProp.regsPerBlock);
    fprintf(outfile, "%*s<WarpSize>%d</WarpSize>\n",
            outLevel, "", curProp.warpSize);
    fprintf(outfile, "%*s<MemPitch>%d</MemPitch>\n",
            outLevel, "", curProp.memPitch);
    fprintf(outfile, "%*s<MaxThreadsPerBlock>%d</MaxThreadsPerBlock>\n",
            outLevel, "", curProp.maxThreadsPerBlock);
    fprintf(outfile, "%*s<MaxThreadsPerDim>\n", outLevel++, "");
    fprintf(outfile, "%*s<X>%d</X>\n", outLevel, "", curProp.maxThreadsDim[0]);
    fprintf(outfile, "%*s<Y>%d</Y>\n", outLevel, "", curProp.maxThreadsDim[1]);
    fprintf(outfile, "%*s<Z>%d</Z>\n", outLevel, "", curProp.maxThreadsDim[2]);
    fprintf(outfile, "%*s</MaxThreadsPerDim>\n", --outLevel, "");
    fprintf(outfile, "%*s<MaxGridSize>\n", outLevel++, "");
    fprintf(outfile, "%*s<X>%d</X>\n", outLevel, "", curProp.maxGridSize[0]);
    fprintf(outfile, "%*s<Y>%d</Y>\n", outLevel, "", curProp.maxGridSize[1]);
    fprintf(outfile, "%*s<Z>%d</Z>\n", outLevel, "", curProp.maxGridSize[2]);
    fprintf(outfile, "%*s</MaxGridSize>\n", --outLevel, "");
    fprintf(outfile, "%*s<ClockRate>%f</ClockRate>\n",
            outLevel, "", curProp.clockRate / 1024.0f / 1024.0f);
    fprintf(outfile, "%*s<TotalConstMem>%d</TotalConstMem>\n",
            outLevel, "", curProp.totalConstMem);
    
    // Print out the other data
    if (gpus[i].numAccessiblePeers > 0) {
      fprintf(outfile, "%*s<AccessiblePeers>\n", outLevel++, "");
      for (j = 0; j < gpus[i].numAccessiblePeers; ++j) {
        fprintf(outfile, "%*s<AccessiblePeer>%d</AccessiblePeer>\n",
                outLevel, "", gpus[i].accessiblePeers[j]);
      }
      fprintf(outfile, "%*s</AccessiblePeers>\n", --outLevel, "");
    }
    fprintf(outfile, "%*s<GlobalMemspace>%d</GlobalMemspace>\n",
            outLevel, "", gpus[i].globalMemspace);
    fprintf(outfile,
            "%*s<TheoreticalPeakGlobalBW>%f</TheoreticalPeakGlobalBW>\n",
            outLevel, "", gpus[i].globalPeakBw);
    
    fprintf(outfile, "%*s</GPU>\n", --outLevel, "");
  }
  
  // Print out Memspace info
  for (i = 0; i < numDevices + GPUOFFSET; ++i) {
    fprintf(outfile, "%*s<MemSpace>\n", outLevel++, "");
    fprintf(outfile, "%*s<Id>%d</Id>\n", outLevel, "", memspaces[i].id);
    fprintf(outfile, "%*s<Bytes>%ld</Bytes>\n", outLevel, "",
            memspaces[i].bytes);
    fprintf(outfile, "%*s</MemSpace>\n", --outLevel, "");
  }
  
  // Print out Memory Transfer B/W info
  for (i = 0; i < numBws; ++i) {
    fprintf(outfile, "%*s<MemXferBW>\n", outLevel++, "");
    fprintf(outfile, "%*s<SrcId>%d</SrcId>\n", outLevel, "", bws[i].srcId);
    fprintf(outfile, "%*s<DstIds>\n", outLevel++, "");
    for (j = 0; j < bws[i].numDsts; ++j) {
      fprintf(outfile, "%*s<DstId>%d</DstId>\n", outLevel, "",
              bws[i].dstIds[j]);
    }
    fprintf(outfile, "%*s</DstIds>\n", --outLevel, "");
    fprintf(outfile, "%*s<BW>%f</BW>\n", outLevel, "", bws[i].bw);
    fprintf(outfile, "%*s</MemXferBW>\n", --outLevel, "");
  }

  fprintf(outfile, "</Machine>\n");
  fclose(outfile);

  // Free memory and return
  free(ram_data);
  hipFree(pinned_data);
  for (i = 0; i < numDevices; ++i) {
    free(gpus[i].accessiblePeers);
    hipFree(dev_datas[i]);
  }
  free(gpus);
  free_xfer_bws(bws, numBws);

  return 0;
}
