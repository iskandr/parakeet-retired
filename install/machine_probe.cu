/*
 * Parakeet
 *
 * (c) 2009-2011 Eric Hielscher, Alex Rubinsteyn
 *
 * Machine Probe
 *
 * Utility for detecting main architectural characteristics of the given
 * computer for use in Parakeet's code optimization.
 * 
 * Outputs an XML file with the gathered information for use by the Parakeet
 * runtime.
 */


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

typedef struct {
  int id;
  hipDeviceProp_t deviceProp;
  int *accessiblePeers;
  int numAccessiblePeers;
  int globalMemspace;
  float globalBw;
} gpu_t;

typedef struct {
  int id;
  uint64_t bytes;
} memspace_t;

typedef struct {
  int srcId;
  int *dstIds;
  int numDsts;
  float bw;
} mem_xfer_bw_t;

void add_xfer_bw(mem_xfer_bw_t **bws, int numBws, int numDsts) {
  mem_xfer_bw_t *tmp =
      (mem_xfer_bw_t*)malloc((numBws + 1) * sizeof(mem_xfer_bw_t));

  if (*bws) {
    memcpy(tmp, *bws, numBws * sizeof(mem_xfer_bw_t));
    free(*bws);
  }

  *bws = tmp;
  (*bws)[numBws].dstIds = (int*)malloc(numDsts * sizeof(int));
  (*bws)[numBws].numDsts = numDsts;
}

void free_xfer_bws(mem_xfer_bw_t *bws, int numBws) {
  if (!bws) return;
  
  int i;
  for (i = 0; i < numBws; ++i) {
    free(bws[i].dstIds);
  }
  free(bws);
}

void chkError(int rslt, char *msg) {
  if (rslt != 0) {
    printf("%s: %d\n", msg, rslt);
    exit(1);
  }
}

double diff_timers(struct timeval *start, struct timeval *end) {
  double ret;

  if (end->tv_usec < start->tv_usec) {
    int nsec = (start->tv_usec - end->tv_usec) / 1000000 + 1;
    start->tv_usec -= 1000000 * nsec;
    start->tv_sec += nsec;
  }
  if (end->tv_usec - start->tv_usec > 1000000) {
    int nsec = (end->tv_usec - start->tv_usec) / 1000000;
    start->tv_usec += 1000000 * nsec;
    start->tv_sec -= nsec;
  }

  ret = (end->tv_sec - start->tv_sec) +
        (end->tv_usec - start->tv_usec) / 1000000.0;

  free(start);
  free(end);

  return ret;
}

struct timeval *gettime(void) {
  struct timeval *ret = (struct timeval*)(malloc(sizeof(struct timeval)));
  gettimeofday(ret, NULL);
  return ret;
}

// Fills the input array with the given value
__global__
void memFill(int *dev_data, int data_size, int val) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (idx < data_size) {
    dev_data[idx] = val;
  }
}

// Tests memory read bandwidth
__global__
void memReadBw(int *dev_data, int data_size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  int step2 = step + step;
  int step3 = step2 + step;
  
  int d1 = dev_data[idx];
  int d2 = dev_data[idx + step];
  int d3 = dev_data[idx + step2];
  int d4 = dev_data[idx + step3];
  dev_data[idx] = d4;
  dev_data[idx + step] = d3;
  dev_data[idx + step2] = d2;
  dev_data[idx + step3] = d1;
}

// Assumes that the current GPU device is set
float time_ram_to_gpu_xfer(void *dev_data, void *ram_data, int data_size) {
  struct timeval *start, *end;
  float ram_gpu_bw = 0.0f;
  hipStreamSynchronize(0);
  start = gettime();
  chkError(hipMemcpy(dev_data, ram_data, data_size, hipMemcpyHostToDevice),
           "Couldn't copy data from RAM to GPU");
  hipStreamSynchronize(0);
  end = gettime();
  ram_gpu_bw += diff_timers(start, end);  
  return data_size / ram_gpu_bw / (1 << 30);
}  

int main(int argc, char **argv) {
  const int RAMID = 0;
  const int PINNEDID = 1;
  const int GPUOFFSET = 2;

  struct timeval *start, *end;
  int i, j;

  // Set up program parameters
  // TODO: We assume here that any GPU we're going to use has at least 128MB of
  //       global memory.  This may not actually be the case.  We probably want
  //       to parameterize this so as to scale to any memory size.
  int data_size = (16 << 20) * sizeof(int);
  char *outFilename = "parakeetconf.xml";
  int debug = 1;

  // Process command line args
  
  // Open output file
  FILE *outfile = fopen(outFilename, "w");
  if (!outfile) {
    printf("Couldn't open output file.\n");
    exit(1);
  }

  // Get number of GPU devices
  int numDevices;
  chkError(hipGetDeviceCount(&numDevices), "Couldn't get number of devices");
  
  // Create a gpu_t struct for each device
  gpu_t *gpus = (gpu_t*)malloc(numDevices * sizeof(gpu_t));
  
  // Create memspace structs for RAM and for each device
  memspace_t *memspaces =
      (memspace_t*)malloc((numDevices + 1) * sizeof(memspace_t));
  for (i = 0; i < numDevices + 1; ++i) {
    memspaces[i].id = i;
  }
  
  // Memory transfer structs for each valid memspace group
  int numBws = 0;
  mem_xfer_bw_t *bws = NULL;
  
  // Set up special RAM memspace
  // TODO: This probably is Ubuntu-specific; need to make it general.
  char *cmd = "awk '{if(NR==1){print $2}}' /proc/meminfo";
  FILE *cmdfile = popen(cmd, "r");
  if (!cmdfile) {
    printf("Unable to get RAM info.\n");
    exit(1);
  }
  char buffer[128];
  memset(buffer, 0, 128);
  if (!fgets(buffer, 128, cmdfile)) {
    printf("Unable to read RAM info from /proc/meminfo.\n");
    exit(1);
  }
  memspaces[RAMID].bytes = (uint64_t)atol(buffer);
  if (!memspaces[RAMID].bytes) {
    printf("Unable to convert RAM info to uint64_t.\n");
    exit(1);
  }
  pclose(cmdfile);
  
  // Allocate some memory for doing RAM <-> GPU transfers.
  int *ram_data = (int*)malloc(data_size);
  int *pinned_data;
  chkError(hipHostMalloc(&pinned_data, data_size, hipHostMallocDefault),
           "Couldn't malloc pinned host mem");
  
  // For each device, get the properties we're interested in
  for (i = 0; i < numDevices; ++i) {
    // Current memspace ID is i + 1, since RAM is 0
    int curId = i + GPUOFFSET;
    
    // Get device properties
    // TODO: Do we need to store this? Could just re-query every time.
    chkError(hipGetDeviceProperties(&gpus[i].deviceProp, i),
             "Couldn't get properties for device");
    
    // Allocate some device memory space
    int *dev_data;
    chkError(hipSetDevice(i), "Couldn't switch GPU devices");
    chkError(hipMalloc(&dev_data, data_size), "Couldn't allocate GPU data");
    
    // Test GPU <-> Global memory bw
    // 1. Fill the global memory with a value
    // 2. Read that value
    int blockWidth = gpus[i].deviceProp.maxThreadsDim[0] / 4;
    int numBlocks = data_size / sizeof(int) / blockWidth;
    memFill<<<numBlocks, blockWidth>>>(dev_data, data_size / sizeof(int), 1);
    numBlocks /= 4;
    hipStreamSynchronize(0);
    start = gettime();
    memReadBw<<<numBlocks, blockWidth>>>(dev_data, data_size / sizeof(int));
    hipStreamSynchronize(0);
    end = gettime();
    float global_mem_bw = data_size / diff_timers(start, end) / (1 << 30);
    
    printf("GPU %d Global Memory BW: %f\n", i, global_mem_bw);
    
    // Test RAM <-> GPU bw
    add_xfer_bw(&bws, numBws, 1);
    bws[numBws].srcId = RAMID;
    bws[numBws].dstIds[0] = curId;
    bws[numBws].bw = time_ram_to_gpu_xfer(dev_data, ram_data, data_size);
    numBws++;
    
    printf("RAM to GPU %d BW: %f\n", i, bws[numBws - 1].bw);
    
    // Test Pinned RAM <-> GPU bw
    add_xfer_bw(&bws, numBws, 1);
    bws[numBws].srcId = PINNEDID;
    bws[numBws].dstIds[0] = curId;
    bws[numBws].bw = time_ram_to_gpu_xfer(dev_data, pinned_data, data_size);
    numBws++;
    
    printf("Pinned RAM to GPU %d BW: %f\n", i, bws[numBws - 1].bw);
    
    // TODO: Test RAM <-> GPU + every other GPU's bw

    // Get peer access info
    gpus[i].numAccessiblePeers = 0;
    int canAccessPeer;
    for (j = 0; j < numDevices; ++j) {
      if (i != j) {
        chkError(hipDeviceCanAccessPeer(&canAccessPeer, i, j),
                 "Couldn't get peer access info");
        if (canAccessPeer) {
          gpus[i].numAccessiblePeers++;
          chkError(hipDeviceEnablePeerAccess(j, 0),
                   "Couldn't enable peer access");
        }
      }
    }
    gpus[i].accessiblePeers =
        (int*)malloc(gpus[i].numAccessiblePeers * sizeof(int));
    int cur = 0;
    for (j = 0; j < numDevices; ++j) {
      if (i != j) {
        chkError(hipDeviceCanAccessPeer(&canAccessPeer, i, j),
                 "Couldn't get peer access info");
        if (canAccessPeer) {
          gpus[i].accessiblePeers[cur++] = j;
          
          // Test P2P memory bandwidth and record 
          chkError(hipSetDevice(j), "Coudln't switch GPU devices");
          int *src_data;
          chkError(hipMalloc(&src_data, data_size),
                   "Couldn't allocate peer GPU data");
          chkError(hipSetDevice(i), "Couldn't switch GPU devices");
          
          float peer_bw = 0.0f;
          hipStreamSynchronize(0);
          start = gettime();
          chkError(hipMemcpyPeer(dev_data, i, src_data, j, data_size),
                    "Couldn't copy data between peer devices");
          hipStreamSynchronize(0);
          end = gettime();
          peer_bw = data_size / diff_timers(start, end) / (1 << 30);
          add_xfer_bw(&bws, numBws, 1);
          bws[numBws].srcId = curId;
          bws[numBws].dstIds[0] = j + GPUOFFSET;
          bws[numBws].bw = peer_bw;
          numBws++;
          
          printf("P2P transfer from %d to %d: %f\n", j, i, peer_bw);
          
          chkError(hipSetDevice(j), "Couldn't switch GPU devices");
          chkError(hipFree(src_data), "Couldn't free peer GPU data");
          chkError(hipSetDevice(i), "Couldn't switch GPU devices");
        }
      }
    }
    
    chkError(hipFree(dev_data), "Couldn't free GPU data");
  }

  free(ram_data);
  hipFree(pinned_data);
  for (i = 0; i < numDevices; ++i) {
    free(gpus[i].accessiblePeers);
  }
  free(gpus);
  free_xfer_bws(bws, numBws);

  return 0;
}
