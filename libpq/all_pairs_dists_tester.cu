#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "all_pairs_dists_1dtex_kernel.cu"
#include "all_pairs_dists_notex_kernel.cu"
#include "base.h"

void usage(void) {
  printf("Usage:\n"
         "--------"
         "-x n       : set number of vectors\n"
         "-y n       : set number of centroids\n"
         "-l n       : set vector length\n"
         "-d         : do 1D\n"
         "-m         : include memory transfer in timing\n"
         );
}

int main(int argc, char **argv) {
  int x_len = 10000;
  int c_len = 128;
  int vec_len = 128;
  int include_mem = 0;
  int doid = 0;
  int i;

  // Process command line
  for (i = 1; i < argc; ++i) {
    if (!strcmp(argv[i], "-x")) {
      if (argc < i + 2) {
        usage();
        exit(-1);
      }
      i += 1;
      x_len = atoi(argv[i]);
    } else if (!strcmp(argv[i], "-y")) {
      if (argc < i + 2) {
        usage();
        exit(-1);
      }
      i += 1;
      c_len = atoi(argv[i]);
    } else if (!strcmp(argv[i], "-l")) {
      if (argc < i + 2) {
        usage();
        exit(-1);
      }
      i += 1;
      vec_len = atoi(argv[i]);
    } else if (!strcmp(argv[i], "-d")) {
      doid = 1;
    } else if (!strcmp(argv[i], "-m")) {
      include_mem = 1;
    }
  }

  int *X;
  int *C;
  float *output;
  hipError_t rslt;
  printf("Allocating host mem\n");
  rslt = hipHostAlloc((void**)&X, x_len * vec_len * sizeof(int),
                       hipHostMallocPortable);
  if (rslt != 0) printf("failed to malloc X: %d\n", rslt);
  rslt = hipHostAlloc((void**)&C, c_len * vec_len * sizeof(int),
                       hipHostMallocPortable);
  if (rslt != 0) printf("failed to malloc C: %d\n", rslt);
  rslt = hipHostAlloc((void**)&output, x_len * c_len * sizeof(float),
                       hipHostMallocPortable);
  if (rslt != 0) printf("failed to malloc output: %d\n", rslt);

  // Initialize array
  printf("initing host input\n");
  for (i = 0; i < x_len * vec_len; ++i) {
    X[i] = rand() % 10;
  }
  for (i = 0; i < c_len * vec_len; ++i) {
    C[i] = rand() % 10;
  }

  float t;
  struct timeval *pqst, *pqe;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *devX;
  int *devC;
  float *devOut;
  int same = 1;
  int j, k;
  float intermediate, result;
  float tol = 0.01f;

    printf("Processing %d vectors, %d centroids, with length %d\n",
          x_len, c_len, vec_len);
    printf("Running 1D texture version\n");

    if (include_mem) {
      printf("Including Memory in xFer & allocation in time\n");
      hipEventRecord(start, 0);
      pqst = pq_gettime();
    }

    rslt = hipMalloc((void**)&devX, x_len * vec_len * sizeof(int));
    check_err(rslt, "Unable to malloc devX");
    rslt = hipMalloc((void**)&devC, c_len * vec_len * sizeof(int));
    check_err(rslt, "Unable to malloc devC");
    rslt = hipMalloc((void**)&devOut, x_len * c_len * sizeof(float));
    check_err(rslt, "Unable to malloc devOut");
    rslt = hipMemcpy(devX, X, x_len * vec_len * sizeof(int),
                      hipMemcpyHostToDevice);
    check_err(rslt, "Unable to copy X to device");
    rslt = hipMemcpy(devC, C, c_len * vec_len * sizeof(int),
                      hipMemcpyHostToDevice);
    check_err(rslt, "Unable to copy C to device");

    if (!include_mem) {
      hipEventRecord(start, 0);
      pqst = pq_gettime();
    }

    hipChannelFormatDesc OneDXDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    hipChannelFormatDesc OneDCDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    rslt = hipBindTexture(0, allDistsLeft1DTex, devX, OneDXDesc,
                          x_len * vec_len * sizeof(int));
    check_err(rslt, "Unable to bind 1D X texture");
    rslt = hipBindTexture(0, allDistsRight1DTex, devC, OneDCDesc,
                          c_len * vec_len * sizeof(int));
    check_err(rslt, "Unable to bind 1D C texture");

    dim3 dim1DBlock(THREADS_PER_DIM, THREADS_PER_DIM);
    dim3 dim1DGrid(safe_div(c_len, THREADS_PER_DIM),
                  safe_div(x_len, THREADS_PER_DIM));

    all_pairs_dists_1dtex_kernel<<<dim1DGrid, dim1DBlock>>>
      (x_len, c_len, vec_len, devOut);

    if (!include_mem) {
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      pqe = pq_gettime();
    }

    rslt = hipMemcpy(output, devOut, x_len * c_len * sizeof(float),
                      hipMemcpyDeviceToHost);
    check_err(rslt, "Error copying 1D output to host");

    if (include_mem) {
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      pqe = pq_gettime();
    }

    hipUnbindTexture(allDistsLeft1DTex);
    hipUnbindTexture(allDistsRight1DTex);

    hipEventElapsedTime(&t, start, stop);
    printf("Wall time for 1D texture: %fms\n", 1000*pq_diff_timers(pqst, pqe));
    printf("GPU time for 1D texture: %fms\n", t);

    // Check validity
    printf("checking validity for 1D case\n");
    for (i = 0; i < x_len; ++i) {
      for (j = 0; j < c_len; ++j) {
        result = 0.0f;
        for (k = 0; k < vec_len; ++k) {
          intermediate = X[i*vec_len + k] - C[j*vec_len + k];
          result += intermediate * intermediate;
        }
        result = sqrt(result);
        if (same && ((fabs(result - output[i*c_len + j]) / result) > tol)) {
          same = 0;
          printf("Different output at X %d, C %d: %f, %f\n", i, j, result,
                output[i*c_len + j]);
        }
      }
    }

    if (same) printf("1D Same!\n");

    hipFree(devX);
    hipFree(devC);
    hipFree(devOut);
}