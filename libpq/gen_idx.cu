#include "hip/hip_runtime.h"
/*
 * Parallel Q Library
 *
 * (c) 2009-2010 Eric Hielscher
 *
 * Each-left each-right implementations.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "base.h"

#define THREADS_PER_BLOCK 256

#ifdef __cplusplus
extern "C" {
#endif

texture<int, 1, hipReadModeElementType> inputTex;
__global__
void gen_index_kernel(int num_input, int *output) {
  int tid = threadIdx.x;
  int id = blockIdx.y * 16384 * THREADS_PER_BLOCK +
           blockIdx.x * THREADS_PER_BLOCK + tid;

  int prev = 0;
  if (id < num_input) {
    if (id > 0) {
      prev = tex1Dfetch(inputTex, id-1);
    }
    
    if (prev != tex1Dfetch(inputTex, id)) {
      output[prev] = id;
    }
  }
}

void launch_gen_index(int *input, int num_input, int *output) {
  hipError_t rslt;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Invoke kernel
  dim3 dimBlock(THREADS_PER_BLOCK);
  int num_blocks = safe_div(num_input, THREADS_PER_BLOCK);
  int gridX, gridY;
  if (num_blocks > 16384) {
    gridX = 16384;
    gridY = safe_div(num_blocks, 16384);
  } else {
    gridX = num_blocks;
    gridY = 1;
  }
  dim3 dimGrid(gridX, gridY);
  printf("grid.x, grid.y: %d %d\n", dimGrid.x, dimGrid.y);

  // Alloc device copies
  int *devInput;
  int *devOutput;
  rslt = hipMalloc((void**)&devInput, num_input * sizeof(int));
  if (rslt != 0) printf("failed to malloc dev Input: %d\n", rslt);
  // Copy data to device
  rslt = hipMemcpy(devInput, input, num_input * sizeof(int),
                    hipMemcpyHostToDevice);
  if (rslt != 0) printf("failed to copy input: %d\n", rslt);
  rslt = hipMalloc((void**)&devOutput, input[num_input-1] * sizeof(int));
  if (rslt != 0) printf("failed to malloc dev Output: %d\n", rslt);

  hipChannelFormatDesc inputDesc =
    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
  inputTex.normalized = 0;
  hipBindTexture(0, inputTex, devInput, inputDesc, num_input * sizeof(int));

  hipEventRecord(start, 0);
  gen_index_kernel<<<dimGrid, dimBlock>>> (num_input, devOutput);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  rslt = hipMemcpy(output, devOutput, sizeof(int) * input[num_input-1],
                    hipMemcpyDeviceToHost);
  if (rslt != 0) printf("failed to copy output to host: %d\n", rslt);

  float t;
  hipEventElapsedTime(&t, start, stop);
  printf("Time for kernel: %f\n", t);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(devInput);
  hipFree(devOutput);
}

void launch_gen_index_dev(int *input, int num_input, int *output,
                          int memtime) {
  hipEvent_t start, stop;
  float t;
  if (memtime) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
  }
  
  //Invoke kernel
  dim3 dimBlock(THREADS_PER_BLOCK);
  int num_blocks = safe_div(num_input, THREADS_PER_BLOCK);
  int gridX, gridY;
  if (num_blocks > 16384) {
    gridX = 16384;
    gridY = safe_div(num_blocks, 16384);
  } else {
    gridX = num_blocks;
    gridY = 1;
  }
  dim3 dimGrid(gridX, gridY);
  printf("[gen_idx] grid.x, grid.y: %d %d\n", dimGrid.x, dimGrid.y);

  hipChannelFormatDesc inputDesc =
    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
  inputTex.normalized = 0;
  hipBindTexture(0, inputTex, input, inputDesc, num_input * sizeof(int));

  gen_index_kernel<<<dimGrid, dimBlock>>> (num_input, output);

  if (memtime) {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t, start, stop);
    printf("Time for gen index: %f\n", t);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
}

#ifdef __cplusplus
}
#endif
